#include <hip/hip_runtime.h>
#include <random>
#include <assert.h>
#include <iostream>
#include <cmath>
// #include <timing.cuh>
#include "../include/defs.cuh"
#include "../include/config.h"
#include "../include/cost_generator.h"
#include "../LAP/Hung_lap.cuh"

int main(int argc, char **argv)
{
  Config config = parseArgs(argc, argv);
  printf("\033[0m");
  printf("Welcome ---------------------\n");
  printConfig(config);

  int seed = config.seed;
  int user_n = config.user_n;
  int dev = config.deviceId;
  int nprob = config.tile;
  if (user_n > 512)
  {
    Log(critical, "implementation not ready yet, exiting...");
    exit(-1);
  }

  // typedef unsigned long data;
  // typedef double data;
  typedef float data;
  double time;
  Timer t;
  data *tcosts = new data[nprob * user_n * user_n];
  data *h_costs;
  for (int prob = 0; prob < nprob; prob++)
  {
    data *costs = generate_cost<data>(config, seed + prob);
    memcpy(&tcosts[prob * user_n * user_n], costs, user_n * user_n * sizeof(data));
    if (prob == 0)
      h_costs = costs;
    else
      delete[] costs;
  }

  time = t.elapsed();
  Log(debug, "cost generation time %f s", time);
  t.reset();
  CUDA_RUNTIME(hipSetDevice(dev));
  data *d_tcosts;
  CUDA_RUNTIME(hipMalloc((void **)&d_tcosts, nprob * user_n * user_n * sizeof(data)));
  CUDA_RUNTIME(hipMemcpy(d_tcosts, tcosts, nprob * user_n * user_n * sizeof(data), hipMemcpyDefault));

  /*BLAP<data> *lap = new BLAP<data>(h_costs, user_n, dev);
  time = t.elapsed();
  Log(debug, "BLAP object generated succesfully in %f s", time);
  t.reset();
  lap->solve();
  time = t.elapsed();
  Log(critical, "solve time %f s\n\n", time);
  delete lap;
  memstatus("post deletion");
  TLAP<data> *tlap = new TLAP<data>((uint)nprob, d_tcosts, user_n, dev);
  time = t.elapsed();
  Log(debug, "TLAP object generated succesfully in %f s", time);
  t.reset();
  tlap->solve();
  time = t.elapsed();
  Log(critical, "solve time %f s\n\n", time);
  delete tlap;*/

  // Try the external solve
  int *Drow_ass;
  data *Drow_duals, *Dcol_duals, *Dobj;

  CUDA_RUNTIME(hipMalloc((void **)&Drow_ass, nprob * user_n * sizeof(int)));
  CUDA_RUNTIME(hipMalloc((void **)&Drow_duals, nprob * user_n * sizeof(int)));
  CUDA_RUNTIME(hipMalloc((void **)&Dcol_duals, nprob * user_n * sizeof(int)));
  CUDA_RUNTIME(hipMalloc((void **)&Dobj, nprob * 1 * sizeof(data)));

  TLAP<data> *tlap = new TLAP<data>(nprob, user_n, dev);
  tlap->solve(d_tcosts, Drow_ass, Drow_duals, Dcol_duals, Dobj);

  // printDebugMatrix<data>(d_tcosts, user_n, user_n, "cost matrix");
  // printDebugArray<data>(Drow_duals, user_n, "row duals");
  // printDebugArray<data>(Dcol_duals, user_n, "col duals");
  // printDebugArray<data>(Dobj, nprob, "objectives");
  CUDA_RUNTIME(hipFree(d_tcosts));
  CUDA_RUNTIME(hipFree(Drow_ass));
  CUDA_RUNTIME(hipFree(Drow_duals));
  CUDA_RUNTIME(hipFree(Dcol_duals));
  CUDA_RUNTIME(hipFree(Dobj));

  delete[] h_costs;
}